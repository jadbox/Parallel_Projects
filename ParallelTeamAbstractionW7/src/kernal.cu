/*

Name: Jonathan Dunlap

Course: Introduction to Parallel and Cloud Computing

CRN: 75092

Assignment: Refactor ParallelTeam

Data: 11/19/2013

*/


#include <hip/hip_runtime.h>
#include <stdio.h>



__global__ void kernel(int *array1, int *array2, int *array3)

{

int index = blockIdx.x * blockDim.x + threadIdx.x;

array3[index] = array1[index] + array2[index];

}



extern "C"

void run_kernel()

{

int i, array1[6], array2[6], array3[6], *devarray1, *devarray2, *devarray3;

for(i = 0; i < 6; i++)

{

array1[i] = i;

array2[i] = 3-i;

}



hipMalloc((void**) &devarray1, sizeof(int)*6);

hipMalloc((void**) &devarray2, sizeof(int)*6);

hipMalloc((void**) &devarray3, sizeof(int)*6);



hipMemcpy(devarray1, array1, sizeof(int)*6, hipMemcpyHostToDevice);

hipMemcpy(devarray2, array2, sizeof(int)*6, hipMemcpyHostToDevice);



kernel<<<2, 3>>>(devarray1, devarray2, devarray3);



hipMemcpy(array3, devarray3, sizeof(int)*6, hipMemcpyDeviceToHost);



for(i = 0; i < 6; i++)

{

printf("%d ", array3[i]);

}

printf("\n");



hipFree(devarray1);

hipFree(devarray2);

hipFree(devarray3);

}

