/*

Name: Jonathan Dunlap

Course: Introduction to Parallel and Cloud Computing

CRN: 75092

Assignment: Refactor ParallelTeam

Data: 11/19/2013

*/

#include <stdio.h>

#include <assert.h>
#include <hip/hip_runtime.h>
#include <cstdlib>

__global__ void incrementArrayOnDevice(int *a, int N, int *count)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if( id < N )
    {
        if( a[id] == 3 )
        {
            atomicAdd(count, 1);
        }
    }
}

extern "C"

int run_kernel(int *a_h, int length)

{
    //int *a_h;   // host
    int *a_d;   // device

    int N = length;

    // allocate array on host
    a_h = (int*)malloc(sizeof(int) * N);
    for(int i = 0; i < N; ++i)
        a_h[i] = (i % 3 == 0 ? 3 : 1);

    // allocate arrays on device
    hipMalloc(&a_d, sizeof(int) * N);

    // copy data from host to device
    hipMemcpy(a_d, a_h, sizeof(int) * N, hipMemcpyHostToDevice);

    // do calculation on device
    int blockSize = 512;
    int nBlocks = N / blockSize + (N % blockSize == 0 ? 0 : 1);
    printf("number of blocks: %d\n", nBlocks);

    int count;
    int *devCount;
    hipMalloc(&devCount, sizeof(int));
    hipMemset(devCount, 0, sizeof(int));

    incrementArrayOnDevice<<<nBlocks, blockSize>>> (a_d, N, devCount);
    
    hipMemcpy(&count, devCount, sizeof(int), hipMemcpyDeviceToHost); // retrieve result from device

    free(a_h);
    hipFree(a_d);
    hipFree(devCount);
    
    return count;
}

